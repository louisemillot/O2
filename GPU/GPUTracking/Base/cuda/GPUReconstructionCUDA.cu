#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

/// \file GPUReconstructionCUDA.cu
/// \author David Rohr

#define GPUCA_GPUCODE_HOSTONLY
#include "GPUReconstructionCUDAIncludesHost.h"

#include <hip/hip_runtime_api.h>

#include "GPUReconstructionCUDA.h"
#include "GPUReconstructionCUDAInternals.h"
#include "GPUReconstructionIncludes.h"
#include "GPUParamRTC.h"
#include "GPUReconstructionCUDAHelpers.inc"

#if defined(GPUCA_KERNEL_COMPILE_MODE) && GPUCA_KERNEL_COMPILE_MODE == 1
#include "utils/qGetLdBinarySymbols.h"
#ifndef __HIPCC__ // CUDA
#define PER_KERNEL_OBJECT_EXT _fatbin
#else // HIP
#define PER_KERNEL_OBJECT_EXT _hip_o
#endif
#define GPUCA_KRNL(x_class, ...) QGET_LD_BINARY_SYMBOLS(GPUCA_M_CAT3(cuda_kernel_module_fatbin_krnl_, GPUCA_M_KRNL_NAME(x_class), PER_KERNEL_OBJECT_EXT))
#include "GPUReconstructionKernelList.h"
#undef GPUCA_KRNL
#endif

static constexpr size_t REQUIRE_MIN_MEMORY = 1024L * 1024 * 1024;
static constexpr size_t REQUIRE_MEMORY_RESERVED = 512L * 1024 * 1024;
static constexpr size_t REQUIRE_FREE_MEMORY_RESERVED_PER_SM = 40L * 1024 * 1024;
static constexpr size_t RESERVE_EXTRA_MEM_THRESHOLD = 10L * 1024 * 1024 * 1024;
static constexpr size_t RESERVE_EXTRA_MEM_OFFSET = 1L * 512 * 1024 * 1024;

using namespace o2::gpu;

__global__ void dummyInitKernel(void*) {}

#include "GPUReconstructionIncludesITS.h"

GPUReconstructionCUDABackend::GPUReconstructionCUDABackend(const GPUSettingsDeviceBackend& cfg) : GPUReconstructionDeviceBase(cfg, sizeof(GPUReconstructionDeviceBase))
{
  if (mMaster == nullptr) {
    mInternals = new GPUReconstructionCUDAInternals;
  }
}

GPUReconstructionCUDABackend::~GPUReconstructionCUDABackend()
{
  if (mMaster == nullptr) {
    delete mInternals;
  }
}

static_assert(sizeof(hipError_t) <= sizeof(int64_t) && hipSuccess == 0);
int32_t GPUReconstructionCUDABackend::GPUChkErrInternal(const int64_t error, const char* file, int32_t line) const
{
  return internal::GPUReconstructionCUDAChkErr(error, file, line);
}

GPUReconstructionCUDA::GPUReconstructionCUDA(const GPUSettingsDeviceBackend& cfg) : GPUReconstructionKernels(cfg)
{
  mDeviceBackendSettings.deviceType = DeviceType::CUDA;
#ifndef __HIPCC__ // CUDA
  mRtcSrcExtension = ".cu";
  mRtcBinExtension = ".fatbin";
#else // HIP
  mRtcSrcExtension = ".hip";
  mRtcBinExtension = ".o";
#endif
}

GPUReconstructionCUDA::~GPUReconstructionCUDA()
{
  Exit(); // Make sure we destroy everything (in particular the ITS tracker) before we exit CUDA
}

GPUReconstruction* GPUReconstruction_Create_CUDA(const GPUSettingsDeviceBackend& cfg) { return new GPUReconstructionCUDA(cfg); }

void GPUReconstructionCUDA::GetITSTraits(std::unique_ptr<o2::its::TrackerTraits>* trackerTraits, std::unique_ptr<o2::its::VertexerTraits>* vertexerTraits, std::unique_ptr<o2::its::TimeFrame>* timeFrame)
{
  if (trackerTraits) {
    trackerTraits->reset(new o2::its::TrackerTraitsGPU);
  }
  if (vertexerTraits) {
    vertexerTraits->reset(new o2::its::VertexerTraitsGPU);
  }
  if (timeFrame) {
    timeFrame->reset(new o2::its::gpu::TimeFrameGPU);
  }
}

void GPUReconstructionCUDA::UpdateAutomaticProcessingSettings()
{
  GPUCA_GPUReconstructionUpdateDefaults();
}

int32_t GPUReconstructionCUDA::InitDevice_Runtime()
{
#ifndef __HIPCC__ // CUDA
  constexpr int32_t reqVerMaj = 2;
  constexpr int32_t reqVerMin = 0;
#endif
  if (mProcessingSettings.rtc.enable && mProcessingSettings.rtc.runTest == 2) {
    genAndLoadRTC();
    exit(0);
  }

  if (mMaster == nullptr) {
    hipDeviceProp_t deviceProp;
    int32_t count, bestDevice = -1;
    double bestDeviceSpeed = -1, deviceSpeed;
    if (GPUChkErrI(hipGetDeviceCount(&count))) {
      GPUError("Error getting CUDA Device Count");
      return (1);
    }
    if (mProcessingSettings.debugLevel >= 2) {
      GPUInfo("Available CUDA devices:");
    }
    std::vector<bool> devicesOK(count, false);
    std::vector<size_t> devMemory(count, 0);
    bool contextCreated = false;
    for (int32_t i = 0; i < count; i++) {
      if (mProcessingSettings.debugLevel >= 4) {
        GPUInfo("Examining device %d", i);
      }
      size_t free, total;
#ifndef __HIPCC__ // CUDA
      if (GPUChkErrI(cudaInitDevice(i, 0, 0))) {
#else // HIP
      if (GPUChkErrI(hipSetDevice(i))) {
#endif
        if (mProcessingSettings.debugLevel >= 4) {
          GPUWarning("Couldn't create context for device %d. Skipping it.", i);
        }
        continue;
      }
      contextCreated = true;
      if (GPUChkErrI(hipMemGetInfo(&free, &total))) {
        if (mProcessingSettings.debugLevel >= 4) {
          GPUWarning("Error obtaining CUDA memory info about device %d! Skipping it.", i);
        }
        GPUChkErr(hipDeviceReset());
        continue;
      }
      if (count > 1) {
        GPUChkErr(hipDeviceReset());
        contextCreated = false;
      }
      if (mProcessingSettings.debugLevel >= 4) {
        GPUInfo("Obtained current memory usage for device %d", i);
      }
      if (GPUChkErrI(hipGetDeviceProperties(&deviceProp, i))) {
        continue;
      }
      if (mProcessingSettings.debugLevel >= 4) {
        GPUInfo("Obtained device properties for device %d", i);
      }
      int32_t deviceOK = true;
      [[maybe_unused]] const char* deviceFailure = "";
#ifndef __HIPCC__
      if (deviceProp.major < reqVerMaj || (deviceProp.major == reqVerMaj && deviceProp.minor < reqVerMin)) {
        deviceOK = false;
        deviceFailure = "Too low device revision";
      }
#endif
      if (free < std::max<size_t>(mDeviceMemorySize, REQUIRE_MIN_MEMORY)) {
        deviceOK = false;
        deviceFailure = "Insufficient GPU memory";
      }

      deviceSpeed = (double)deviceProp.multiProcessorCount * (double)deviceProp.clockRate * (double)deviceProp.warpSize * (double)free * (double)deviceProp.major * (double)deviceProp.major;
      if (mProcessingSettings.debugLevel >= 2) {
        GPUImportant("Device %s%2d: %s (Rev: %d.%d - Mem Avail %lu / %lu)%s %s", deviceOK ? " " : "[", i, deviceProp.name, deviceProp.major, deviceProp.minor, free, (size_t)deviceProp.totalGlobalMem, deviceOK ? " " : " ]", deviceOK ? "" : deviceFailure);
      }
      if (!deviceOK) {
        continue;
      }
      devicesOK[i] = true;
      devMemory[i] = std::min<size_t>(free, std::max<int64_t>(0, total - REQUIRE_MEMORY_RESERVED));
      if (deviceSpeed > bestDeviceSpeed) {
        bestDevice = i;
        bestDeviceSpeed = deviceSpeed;
      } else {
        if (mProcessingSettings.debugLevel >= 2 && mProcessingSettings.deviceNum < 0) {
          GPUInfo("Skipping: Speed %f < %f\n", deviceSpeed, bestDeviceSpeed);
        }
      }
    }

    bool noDevice = false;
    if (bestDevice == -1) {
      GPUWarning("No %sCUDA Device available, aborting CUDA Initialisation (Required mem: %ld)", count ? "appropriate " : "", (int64_t)mDeviceMemorySize);
#ifndef __HIPCC__
      GPUImportant("Requiring Revision %d.%d, Mem: %lu", reqVerMaj, reqVerMin, std::max<size_t>(mDeviceMemorySize, REQUIRE_MIN_MEMORY));
#endif
      noDevice = true;
    } else if (mProcessingSettings.deviceNum > -1) {
      if (mProcessingSettings.deviceNum >= (signed)count) {
        GPUError("Requested device ID %d does not exist", mProcessingSettings.deviceNum);
        noDevice = true;
      } else if (!devicesOK[mProcessingSettings.deviceNum]) {
        GPUError("Unsupported device requested (%d)", mProcessingSettings.deviceNum);
        noDevice = true;
      } else {
        bestDevice = mProcessingSettings.deviceNum;
      }
    }
    if (noDevice) {
      if (contextCreated) {
        GPUChkErrI(hipDeviceReset());
      }
      return (1);
    }
    mDeviceId = bestDevice;

    GPUChkErrI(hipGetDeviceProperties(&deviceProp, mDeviceId));

    if (mProcessingSettings.debugLevel >= 2) {
      GPUInfo("Using CUDA Device %s with Properties:", deviceProp.name);
      GPUInfo("\ttotalGlobalMem = %ld", (uint64_t)deviceProp.totalGlobalMem);
      GPUInfo("\tsharedMemPerBlock = %ld", (uint64_t)deviceProp.sharedMemPerBlock);
      GPUInfo("\tregsPerBlock = %d", deviceProp.regsPerBlock);
      GPUInfo("\twarpSize = %d", deviceProp.warpSize);
      GPUInfo("\tmemPitch = %ld", (uint64_t)deviceProp.memPitch);
      GPUInfo("\tmaxThreadsPerBlock = %d", deviceProp.maxThreadsPerBlock);
      GPUInfo("\tmaxThreadsDim = %d %d %d", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
      GPUInfo("\tmaxGridSize = %d %d %d", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
      GPUInfo("\ttotalConstMem = %ld", (uint64_t)deviceProp.totalConstMem);
      GPUInfo("\tmajor = %d", deviceProp.major);
      GPUInfo("\tminor = %d", deviceProp.minor);
      GPUInfo("\tclockRate = %d", deviceProp.clockRate);
      GPUInfo("\tmemoryClockRate = %d", deviceProp.memoryClockRate);
      GPUInfo("\tmultiProcessorCount = %d", deviceProp.multiProcessorCount);
      GPUInfo("\ttextureAlignment = %ld", (uint64_t)deviceProp.textureAlignment);
      GPUInfo(" ");
    }
    if (deviceProp.warpSize != GPUCA_WARP_SIZE) {
      throw std::runtime_error("Invalid warp size on GPU");
    }
    mBlockCount = deviceProp.multiProcessorCount;
    mMaxBackendThreads = std::max<int32_t>(mMaxBackendThreads, deviceProp.maxThreadsPerBlock * mBlockCount);
#ifndef __HIPCC__ // CUDA
    mWarpSize = 32;
#else // HIP
    mWarpSize = 64;
#endif
    mDeviceName = deviceProp.name;
    mDeviceName += " (CUDA GPU)";

    if (deviceProp.major < 3) {
      GPUError("Unsupported CUDA Device");
      return (1);
    }

#ifdef GPUCA_USE_TEXTURES
    if (GPUCA_SECTOR_DATA_MEMORY * NSECTORS > (size_t)deviceProp.maxTexture1DLinear) {
      GPUError("Invalid maximum texture size of device: %ld < %ld\n", (int64_t)deviceProp.maxTexture1DLinear, (int64_t)(GPUCA_SECTOR_DATA_MEMORY * NSECTORS));
      return (1);
    }
#endif
#ifndef GPUCA_NO_CONSTANT_MEMORY
    if (gGPUConstantMemBufferSize > deviceProp.totalConstMem) {
      GPUError("Insufficient constant memory available on GPU %d < %d!", (int32_t)deviceProp.totalConstMem, (int32_t)gGPUConstantMemBufferSize);
      return (1);
    }
#endif

#ifndef __HIPCC__ // CUDA
    if (contextCreated == 0 && GPUChkErrI(cudaInitDevice(mDeviceId, 0, 0))) {
#else // HIP
    if (contextCreated == 0 && GPUChkErrI(hipSetDevice(mDeviceId))) {
#endif
      GPUError("Could not set CUDA Device!");
      return (1);
    }

#ifndef __HIPCC__ // CUDA
    if (GPUChkErrI(hipDeviceSetLimit(hipLimitStackSize, GPUCA_GPU_STACK_SIZE))) {
      GPUError("Error setting CUDA stack size");
      GPUChkErrI(hipDeviceReset());
      return (1);
    }
    if (GPUChkErrI(hipDeviceSetLimit(hipLimitMallocHeapSize, mProcessingSettings.deterministicGPUReconstruction ? std::max<size_t>(1024 * 1024 * 1024, GPUCA_GPU_HEAP_SIZE) : GPUCA_GPU_HEAP_SIZE))) {
      GPUError("Error setting CUDA stack size");
      GPUChkErrI(hipDeviceReset());
      return (1);
    }
#else // HIP
    if (GPUChkErrI(hipSetDeviceFlags(hipDeviceScheduleBlockingSync))) {
      GPUError("Could not set HIP Device flags!");
      return (1);
    }
#endif

    if (mDeviceMemorySize == 1 || mDeviceMemorySize == 2) {
      mDeviceMemorySize = std::max<int64_t>(0, devMemory[mDeviceId] - REQUIRE_FREE_MEMORY_RESERVED_PER_SM * deviceProp.multiProcessorCount); // Take all GPU memory but some reserve
      if (mDeviceMemorySize >= RESERVE_EXTRA_MEM_THRESHOLD) {
        mDeviceMemorySize -= RESERVE_EXTRA_MEM_OFFSET;
      }
    }
    if (mDeviceMemorySize == 2) {
      mDeviceMemorySize = mDeviceMemorySize * 2 / 3; // Leave 1/3 of GPU memory for event display
    }

    if (mProcessingSettings.debugLevel >= 3) {
      GPUInfo("Allocating memory on GPU");
    }
    if (mDeviceMemorySize > deviceProp.totalGlobalMem || GPUChkErrI(hipMalloc(&mDeviceMemoryBase, mDeviceMemorySize))) {
      size_t free, total;
      GPUChkErr(hipMemGetInfo(&free, &total));
      GPUError("CUDA Memory Allocation Error (trying %ld bytes, %ld available on GPU, %ld free)", (int64_t)mDeviceMemorySize, (int64_t)deviceProp.totalGlobalMem, (int64_t)free);
      GPUChkErrI(hipDeviceReset());
      return (1);
    }
    if (mProcessingSettings.debugLevel >= 3) {
      GPUInfo("Allocating memory on Host");
    }
    if (GPUChkErrI(hipHostMalloc(&mHostMemoryBase, mHostMemorySize))) {
      GPUError("Error allocating Page Locked Host Memory (trying %ld bytes)", (int64_t)mHostMemorySize);
      GPUChkErrI(hipDeviceReset());
      return (1);
    }
    if (mProcessingSettings.debugLevel >= 1) {
      GPUInfo("Memory ptrs: GPU (%ld bytes): %p - Host (%ld bytes): %p", (int64_t)mDeviceMemorySize, mDeviceMemoryBase, (int64_t)mHostMemorySize, mHostMemoryBase);
      memset(mHostMemoryBase, 0xDD, mHostMemorySize);
      if (GPUChkErrI(hipMemset(mDeviceMemoryBase, 0xDD, mDeviceMemorySize))) {
        GPUError("Error during CUDA memset");
        GPUChkErrI(hipDeviceReset());
        return (1);
      }
    }

    for (int32_t i = 0; i < mNStreams; i++) {
      if (GPUChkErrI(hipStreamCreateWithFlags(&mInternals->Streams[i], hipStreamNonBlocking))) {
        GPUError("Error creating CUDA Stream");
        GPUChkErrI(hipDeviceReset());
        return (1);
      }
    }

#ifndef __HIPCC__ // CUDA
    dummyInitKernel<<<mBlockCount, 256>>>(mDeviceMemoryBase);
#else // HIP
    hipLaunchKernelGGL(HIP_KERNEL_NAME(dummyInitKernel), dim3(mBlockCount), dim3(256), 0, 0, mDeviceMemoryBase);
#endif

    if (mProcessingSettings.rtc.enable) {
      genAndLoadRTC();
    }
#if defined(GPUCA_KERNEL_COMPILE_MODE) && GPUCA_KERNEL_COMPILE_MODE == 1
    else {
#define GPUCA_KRNL(x_class, ...)                                        \
  mInternals->kernelModules.emplace_back(std::make_unique<hipModule_t>()); \
  GPUChkErr(hipModuleLoadData(mInternals->kernelModules.back().get(), GPUCA_M_CAT3(_binary_cuda_kernel_module_fatbin_krnl_, GPUCA_M_KRNL_NAME(x_class), GPUCA_M_CAT(PER_KERNEL_OBJECT_EXT, _start))));
#include "GPUReconstructionKernelList.h"
#undef GPUCA_KRNL
      loadKernelModules(true);
    }
#endif
    void* devPtrConstantMem = nullptr;
#ifndef GPUCA_NO_CONSTANT_MEMORY
    runConstantRegistrators();
    devPtrConstantMem = mDeviceConstantMemList[0];
    for (uint32_t i = 0; i < mInternals->kernelModules.size(); i++) {
#ifndef __HIPCC__
      hipDeviceptr_t tmp; // CUDA has a custom type, that initializes to zero and cannot be initialized with nullptr
#else
      hipDeviceptr_t tmp = nullptr; // HIP just uses void*
#endif
      size_t tmpSize = 0;
      GPUChkErr(hipModuleGetGlobal(&tmp, &tmpSize, *mInternals->kernelModules[i], "gGPUConstantMemBuffer"));
      mDeviceConstantMemList.emplace_back((void*)tmp);
    }
#else
    GPUChkErr(hipMalloc(&devPtrConstantMem, gGPUConstantMemBufferSize));
#endif
    mDeviceConstantMem = (GPUConstantMem*)devPtrConstantMem;

    GPUInfo("CUDA Initialisation successfull (Device %d: %s (Frequency %d, Cores %d), %ld / %ld bytes host / global memory, Stack frame %d, Constant memory %ld)", mDeviceId, deviceProp.name, deviceProp.clockRate, deviceProp.multiProcessorCount, (int64_t)mHostMemorySize, (int64_t)mDeviceMemorySize, (int32_t)GPUCA_GPU_STACK_SIZE, (int64_t)gGPUConstantMemBufferSize);
  } else {
    GPUReconstructionCUDA* master = dynamic_cast<GPUReconstructionCUDA*>(mMaster);
    mDeviceId = master->mDeviceId;
    mBlockCount = master->mBlockCount;
    mWarpSize = master->mWarpSize;
    mMaxBackendThreads = master->mMaxBackendThreads;
    mDeviceName = master->mDeviceName;
    mDeviceConstantMem = master->mDeviceConstantMem;
    mDeviceConstantMemList.resize(master->mDeviceConstantMemList.size());
    std::copy(master->mDeviceConstantMemList.begin(), master->mDeviceConstantMemList.end(), mDeviceConstantMemList.begin());
    mInternals = master->mInternals;
    GPUChkErr(hipSetDevice(mDeviceId));

    GPUInfo("CUDA Initialisation successfull (from master)");
  }

  for (uint32_t i = 0; i < mEvents.size(); i++) {
    hipEvent_t* events = (hipEvent_t*)mEvents[i].data();
    for (uint32_t j = 0; j < mEvents[i].size(); j++) {
#ifndef __HIPCC__ // CUDA
      if (GPUChkErrI(hipEventCreate(&events[j]))) {
#else
      if (GPUChkErrI(hipEventCreateWithFlags(&events[j], hipEventBlockingSync))) {
#endif
        GPUError("Error creating event");
        GPUChkErrI(hipDeviceReset());
        return 1;
      }
    }
  }

  return (0);
}

void GPUReconstructionCUDA::genAndLoadRTC()
{
  std::string filename = "";
  uint32_t nCompile = 0;
  if (genRTC(filename, nCompile)) {
    throw std::runtime_error("Runtime compilation failed");
  }
  for (uint32_t i = 0; i < nCompile; i++) {
    if (mProcessingSettings.rtc.runTest != 2) {
      mInternals->kernelModules.emplace_back(std::make_unique<hipModule_t>());
      GPUChkErr(hipModuleLoad(mInternals->kernelModules.back().get(), (filename + "_" + std::to_string(i) + mRtcBinExtension).c_str()));
    }
    remove((filename + "_" + std::to_string(i) + mRtcSrcExtension).c_str());
    remove((filename + "_" + std::to_string(i) + mRtcBinExtension).c_str());
  }
  if (mProcessingSettings.rtc.runTest == 2) {
    return;
  }
  loadKernelModules(mProcessingSettings.rtc.compilePerKernel);
}

int32_t GPUReconstructionCUDA::ExitDevice_Runtime()
{
  // Uninitialize CUDA
  GPUChkErr(hipSetDevice(mDeviceId));
  SynchronizeGPU();
  unregisterRemainingRegisteredMemory();

  for (uint32_t i = 0; i < mEvents.size(); i++) {
    hipEvent_t* events = (hipEvent_t*)mEvents[i].data();
    for (uint32_t j = 0; j < mEvents[i].size(); j++) {
      GPUChkErrI(hipEventDestroy(events[j]));
    }
  }

  if (mMaster == nullptr) {
    GPUChkErrI(hipFree(mDeviceMemoryBase));
#ifdef GPUCA_NO_CONSTANT_MEMORY
    GPUChkErrI(hipFree(mDeviceConstantMem));
#endif

    for (int32_t i = 0; i < mNStreams; i++) {
      GPUChkErrI(hipStreamDestroy(mInternals->Streams[i]));
    }

    GPUChkErrI(hipHostFree(mHostMemoryBase));
    for (uint32_t i = 0; i < mInternals->kernelModules.size(); i++) {
      GPUChkErr(hipModuleUnload(*mInternals->kernelModules[i]));
    }

    GPUChkErrI(hipDeviceReset());
    GPUInfo("CUDA Uninitialized");
  }
  mDeviceMemoryBase = nullptr;
  mHostMemoryBase = nullptr;

  return (0);
}

size_t GPUReconstructionCUDA::GPUMemCpy(void* dst, const void* src, size_t size, int32_t stream, int32_t toGPU, deviceEvent* ev, deviceEvent* evList, int32_t nEvents)
{
  if (mProcessingSettings.debugLevel >= 3) {
    stream = -1;
  }
  if (stream == -1) {
    SynchronizeGPU();
    GPUChkErr(hipMemcpy(dst, src, size, toGPU ? hipMemcpyHostToDevice : hipMemcpyDeviceToHost));
  } else {
    if (evList == nullptr) {
      nEvents = 0;
    }
    for (int32_t k = 0; k < nEvents; k++) {
      GPUChkErr(hipStreamWaitEvent(mInternals->Streams[stream], evList[k].get<hipEvent_t>(), 0));
    }
    GPUChkErr(hipMemcpyAsync(dst, src, size, toGPU == -2 ? hipMemcpyDeviceToDevice : (toGPU ? hipMemcpyHostToDevice : hipMemcpyDeviceToHost), mInternals->Streams[stream]));
  }
  if (ev) {
    GPUChkErr(hipEventRecord(ev->get<hipEvent_t>(), mInternals->Streams[stream == -1 ? 0 : stream]));
  }
  if (mProcessingSettings.serializeGPU & 2) {
    GPUDebug(("GPUMemCpy " + std::to_string(toGPU)).c_str(), stream, true);
  }
  return size;
}

size_t GPUReconstructionCUDA::WriteToConstantMemory(size_t offset, const void* src, size_t size, int32_t stream, deviceEvent* ev)
{
  for (uint32_t i = 0; i < 1 + mDeviceConstantMemList.size(); i++) {
    void* basePtr = i ? mDeviceConstantMemList[i - 1] : mDeviceConstantMem;
    if (basePtr == nullptr || (i && basePtr == (void*)mDeviceConstantMem)) {
      continue;
    }
    if (stream == -1) {
      GPUChkErr(hipMemcpy(((char*)basePtr) + offset, src, size, hipMemcpyHostToDevice));
    } else {
      GPUChkErr(hipMemcpyAsync(((char*)basePtr) + offset, src, size, hipMemcpyHostToDevice, mInternals->Streams[stream]));
    }
  }
  if (ev && stream != -1) {
    GPUChkErr(hipEventRecord(ev->get<hipEvent_t>(), mInternals->Streams[stream]));
  }
  if (mProcessingSettings.serializeGPU & 2) {
    GPUDebug("WriteToConstantMemory", stream, true);
  }
  return size;
}

void GPUReconstructionCUDA::ReleaseEvent(deviceEvent ev) {}
void GPUReconstructionCUDA::RecordMarker(deviceEvent* ev, int32_t stream) { GPUChkErr(hipEventRecord(ev->get<hipEvent_t>(), mInternals->Streams[stream])); }

std::unique_ptr<gpu_reconstruction_kernels::threadContext> GPUReconstructionCUDA::GetThreadContext()
{
  GPUChkErr(hipSetDevice(mDeviceId));
  return GPUReconstructionProcessing::GetThreadContext();
}

void GPUReconstructionCUDA::SynchronizeGPU() { GPUChkErr(hipDeviceSynchronize()); }
void GPUReconstructionCUDA::SynchronizeStream(int32_t stream) { GPUChkErr(hipStreamSynchronize(mInternals->Streams[stream])); }

void GPUReconstructionCUDA::SynchronizeEvents(deviceEvent* evList, int32_t nEvents)
{
  for (int32_t i = 0; i < nEvents; i++) {
    GPUChkErr(hipEventSynchronize(evList[i].get<hipEvent_t>()));
  }
}

void GPUReconstructionCUDA::StreamWaitForEvents(int32_t stream, deviceEvent* evList, int32_t nEvents)
{
  for (int32_t i = 0; i < nEvents; i++) {
    GPUChkErr(hipStreamWaitEvent(mInternals->Streams[stream], evList[i].get<hipEvent_t>(), 0));
  }
}

bool GPUReconstructionCUDA::IsEventDone(deviceEvent* evList, int32_t nEvents)
{
  for (int32_t i = 0; i < nEvents; i++) {
    hipError_t retVal = hipEventSynchronize(evList[i].get<hipEvent_t>());
    if (retVal == hipErrorNotReady) {
      return false;
    }
    GPUChkErr(retVal);
  }
  return (true);
}

int32_t GPUReconstructionCUDA::GPUDebug(const char* state, int32_t stream, bool force)
{
  // Wait for CUDA-Kernel to finish and check for CUDA errors afterwards, in case of debugmode
  hipError_t cuErr;
  cuErr = hipGetLastError();
  if (cuErr != hipSuccess) {
    GPUError("CUDA Error %s while running (%s) (Stream %d)", hipGetErrorString(cuErr), state, stream);
    return (1);
  }
  if (!force && mProcessingSettings.debugLevel <= 0) {
    return (0);
  }
  if (GPUChkErrI(stream == -1 ? hipDeviceSynchronize() : hipStreamSynchronize(mInternals->Streams[stream]))) {
    GPUError("CUDA Error while synchronizing (%s) (Stream %d)", state, stream);
    return (1);
  }
  if (mProcessingSettings.debugLevel >= 3) {
    GPUInfo("GPU Sync Done");
  }
  return (0);
}

int32_t GPUReconstructionCUDA::registerMemoryForGPU_internal(const void* ptr, size_t size)
{
  if (mProcessingSettings.debugLevel >= 3) {
    GPUInfo("Registering %zu bytes of memory for GPU", size);
  }
  return GPUChkErrI(hipHostRegister((void*)ptr, size, hipHostRegisterDefault));
}

int32_t GPUReconstructionCUDA::unregisterMemoryForGPU_internal(const void* ptr)
{
  return GPUChkErrI(hipHostUnregister((void*)ptr));
}

void GPUReconstructionCUDABackend::PrintKernelOccupancies()
{
  int32_t maxBlocks = 0, threads = 0, suggestedBlocks = 0, nRegs = 0, sMem = 0;
  GPUChkErr(hipSetDevice(mDeviceId));
  for (uint32_t i = 0; i < mInternals->kernelFunctions.size(); i++) {
    GPUChkErr(hipModuleOccupancyMaxPotentialBlockSize(&suggestedBlocks, &threads, *mInternals->kernelFunctions[i], 0, 0)); // NOLINT: failure in clang-tidy
    GPUChkErr(hipModuleOccupancyMaxActiveBlocksPerMultiprocessor(&maxBlocks, *mInternals->kernelFunctions[i], threads, 0));
    GPUChkErr(hipFuncGetAttribute(&nRegs, HIP_FUNC_ATTRIBUTE_NUM_REGS, *mInternals->kernelFunctions[i]));
    GPUChkErr(hipFuncGetAttribute(&sMem, HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, *mInternals->kernelFunctions[i]));
    GPUInfo("Kernel: %50s Block size: %4d, Maximum active blocks: %3d, Suggested blocks: %3d, Regs: %3d, smem: %3d", GetKernelName(i).c_str(), threads, maxBlocks, suggestedBlocks, nRegs, sMem);
  }
}

void GPUReconstructionCUDA::loadKernelModules(bool perKernel)
{
  uint32_t j = 0;
#define GPUCA_KRNL(x_class, ...)                                                                                                                                                        \
  if (GetKernelNum<GPUCA_M_KRNL_TEMPLATE(x_class)>() != j) {                                                                                                                            \
    GPUFatal("kernel numbers out of sync");                                                                                                                                             \
  }                                                                                                                                                                                     \
  mInternals->kernelFunctions.emplace_back(new hipFunction_t);                                                                                                                             \
  if (mProcessingSettings.debugLevel >= 3) {                                                                                                                                            \
    GPUInfo("Loading kernel %s (j = %u)", GPUCA_M_STR(GPUCA_M_CAT(krnl_, GPUCA_M_KRNL_NAME(x_class))), j);                                                                              \
  }                                                                                                                                                                                     \
  GPUChkErr(hipModuleGetFunction(mInternals->kernelFunctions.back().get(), *mInternals->kernelModules[perKernel ? j : 0], GPUCA_M_STR(GPUCA_M_CAT(krnl_, GPUCA_M_KRNL_NAME(x_class))))); \
  j++;
#include "GPUReconstructionKernelList.h"
#undef GPUCA_KRNL
  if (j != mInternals->kernelModules.size()) {
    GPUFatal("Did not load all kernels (%u < %u)", j, (uint32_t)mInternals->kernelModules.size());
  }
}

#ifndef __HIPCC__ // CUDA
int32_t GPUReconstructionCUDA::PrepareTextures()
{
#ifdef GPUCA_USE_TEXTURES
  hipChannelFormatDesc channelDescu2 = hipCreateChannelDesc<cahit2>();
  size_t offset;
  GPUChkErr(hipBindTexture(&offset, &gAliTexRefu2, mProcessorsShadow->tpcTrackers[0].Data().Memory(), &channelDescu2, NSECTORS * GPUCA_SECTOR_DATA_MEMORY));
  hipChannelFormatDesc channelDescu = hipCreateChannelDesc<calink>();
  GPUChkErr(hipBindTexture(&offset, &gAliTexRefu, mProcessorsShadow->tpcTrackers[0].Data().Memory(), &channelDescu, NSECTORS * GPUCA_SECTOR_DATA_MEMORY));
#endif
  return (0);
}

void GPUReconstructionCUDA::startGPUProfiling()
{
  GPUChkErr(hipProfilerStart());
}

void GPUReconstructionCUDA::endGPUProfiling()
{
  GPUChkErr(hipProfilerStop());
}
#else  // HIP
void* GPUReconstructionHIP::getGPUPointer(void* ptr)
{
  void* retVal = nullptr;
  GPUChkErr(hipHostGetDevicePointer(&retVal, ptr, 0));
  return retVal;
}
#endif // __HIPCC__

namespace o2::gpu
{
template class GPUReconstructionKernels<GPUReconstructionCUDABackend>;
}
