#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

/// \file GPUReconstructionCUDAKernels.cu
/// \author David Rohr

#include "GPUReconstructionCUDAIncludesHost.h"

#include "GPUReconstructionCUDA.h"
#include "GPUReconstructionCUDAInternals.h"

using namespace o2::gpu;

#ifdef GPUCA_USE_TEXTURES
texture<cahit2, hipTextureType1D, hipReadModeElementType> gAliTexRefu2;
texture<calink, hipTextureType1D, hipReadModeElementType> gAliTexRefu;
#endif

#include "GPUReconstructionIncludesDeviceAll.h"

#if defined(__HIPCC__) && defined(GPUCA_HAS_GLOBAL_SYMBOL_CONSTANT_MEM)
__global__ void gGPUConstantMemBuffer_dummy(int32_t* p) { *p = *(int32_t*)&gGPUConstantMemBuffer; }
#endif

template <>
inline void GPUReconstructionCUDABackend::runKernelBackendInternal<GPUMemClean16, 0>(const krnlSetupTime& _xyz, void* const& ptr, uint64_t const& size)
{
  GPUChkErr(hipMemsetAsync(ptr, 0, size, mInternals->Streams[_xyz.x.stream]));
}

template <class T, int32_t I, typename... Args>
inline void GPUReconstructionCUDABackend::runKernelBackendInternal(const krnlSetupTime& _xyz, const Args&... args)
{
#if !defined(GPUCA_KERNEL_COMPILE_MODE) || GPUCA_KERNEL_COMPILE_MODE != 1
  if (!mProcessingSettings.rtc.enable) {
    backendInternal<T, I>::runKernelBackendMacro(_xyz, this, args...);
  } else
#endif
  {
    auto& x = _xyz.x;
    auto& y = _xyz.y;
    const void* pArgs[sizeof...(Args) + 3]; // 3 is max: cons mem + y.index + y.num
    int32_t arg_offset = 0;
#ifdef GPUCA_NO_CONSTANT_MEMORY
    arg_offset = 1;
    pArgs[0] = &mDeviceConstantMem;
#endif
    pArgs[arg_offset] = &y.index;
    GPUReconstructionCUDAInternals::getArgPtrs(&pArgs[arg_offset + 1], args...);
    GPUChkErr(hipModuleLaunchKernel(*mInternals->kernelFunctions[getRTCkernelNum<false, T, I>()], x.nBlocks, 1, 1, x.nThreads, 1, 1, 0, mInternals->Streams[x.stream], (void**)pArgs, nullptr));
  }
}

template <class T, int32_t I, typename... Args>
void GPUReconstructionCUDABackend::runKernelBackend(const krnlSetupArgs<T, I, Args...>& args)
{
  auto& x = args.s.x;
  auto& z = args.s.z;
  if (z.evList) {
    for (int32_t k = 0; k < z.nEvents; k++) {
      GPUChkErr(hipStreamWaitEvent(mInternals->Streams[x.stream], ((hipEvent_t*)z.evList)[k], 0));
    }
  }
  {
    GPUDebugTiming timer(mProcessingSettings.deviceTimers && mProcessingSettings.debugLevel > 0, (deviceEvent*)mDebugEvents, mInternals->Streams, args.s, this);
    std::apply([this, &args](auto&... vals) { this->runKernelBackendInternal<T, I, Args...>(args.s, vals...); }, args.v);
  }
  GPUChkErr(hipGetLastError());
  if (z.ev) {
    GPUChkErr(hipEventRecord(*(hipEvent_t*)z.ev, mInternals->Streams[x.stream]));
  }
}

#undef GPUCA_KRNL_REG
#define GPUCA_KRNL_REG(args) __launch_bounds__(GPUCA_M_MAX2_3(GPUCA_M_STRIP(args)))

#if defined(GPUCA_KERNEL_COMPILE_MODE) && GPUCA_KERNEL_COMPILE_MODE == 1 // ---------- COMPILE_MODE = perkernel ----------
#define GPUCA_KRNL(x_class, x_attributes, x_arguments, x_forward, x_types) \
  GPUCA_KRNL_PROP(x_class, x_attributes)                                   \
  template void GPUReconstructionCUDABackend::runKernelBackend<GPUCA_M_KRNL_TEMPLATE(x_class)>(const krnlSetupArgs<GPUCA_M_KRNL_TEMPLATE(x_class) GPUCA_M_STRIP(x_types)>& args);
#else // ---------- COMPILE_MODE = onefile | rdc ----------
#if defined(GPUCA_KERNEL_COMPILE_MODE) && GPUCA_KERNEL_COMPILE_MODE == 2
#define GPUCA_KRNL_DEFONLY // COMPILE_MODE = rdc
#endif

#define GPUCA_KRNL(x_class, x_attributes, x_arguments, x_forward, x_types) \
  GPUCA_KRNL_PROP(x_class, x_attributes)                                   \
  GPUCA_KRNL_HOST(x_class, x_attributes, x_arguments, x_forward, x_types)  \
  template void GPUReconstructionCUDABackend::runKernelBackend<GPUCA_M_KRNL_TEMPLATE(x_class)>(const krnlSetupArgs<GPUCA_M_KRNL_TEMPLATE(x_class) GPUCA_M_STRIP(x_types)>& args);

#ifndef __HIPCC__ // CUDA version
#define GPUCA_KRNL_CALL(x_class, ...) \
  GPUCA_M_CAT(krnl_, GPUCA_M_KRNL_NAME(x_class))<<<x.nBlocks, x.nThreads, 0, me->mInternals->Streams[x.stream]>>>(GPUCA_CONSMEM_CALL y.index, args...);
#else // HIP version
#undef GPUCA_KRNL_CUSTOM
#define GPUCA_KRNL_CUSTOM(args) GPUCA_M_STRIP(args)
#define GPUCA_KRNL_CALL(x_class, ...) \
  hipLaunchKernelGGL(HIP_KERNEL_NAME(GPUCA_M_CAT(krnl_, GPUCA_M_KRNL_NAME(x_class))), dim3(x.nBlocks), dim3(x.nThreads), 0, me->mInternals->Streams[x.stream], GPUCA_CONSMEM_CALL y.index, args...);
#endif // __HIPCC__

#endif // ---------- COMPILE_MODE = onefile | rdc ----------

#include "GPUReconstructionKernelList.h"
#undef GPUCA_KRNL

template <bool multi, class T, int32_t I>
int32_t GPUReconstructionCUDABackend::getRTCkernelNum(int32_t k)
{
  static int32_t num = k;
  if (num < 0) {
    throw std::runtime_error("Invalid kernel");
  }
  return num;
}

#define GPUCA_KRNL(x_class, ...)                                                                                    \
  template int32_t GPUReconstructionCUDABackend::getRTCkernelNum<false, GPUCA_M_KRNL_TEMPLATE(x_class)>(int32_t k); \
  template int32_t GPUReconstructionCUDABackend::getRTCkernelNum<true, GPUCA_M_KRNL_TEMPLATE(x_class)>(int32_t k);
#include "GPUReconstructionKernelList.h"
#undef GPUCA_KRNL

void GPUReconstructionCUDABackend::getRTCKernelCalls(std::vector<std::string>& kernels)
{
#define GPUCA_KRNL(...) kernels.emplace_back(GPUCA_M_STR(GPUCA_KRNLGPU(__VA_ARGS__)));
#include "GPUReconstructionKernelList.h"
#undef GPUCA_KRNL
}

#ifndef GPUCA_NO_CONSTANT_MEMORY
static GPUReconstructionDeviceBase::deviceConstantMemRegistration registerConstSymbol([]() {
  void* retVal = nullptr;
  if (GPUChkErrS(hipGetSymbolAddress(&retVal, HIP_SYMBOL(gGPUConstantMemBuffer)))) {
    throw std::runtime_error("Could not obtain GPU constant memory symbol");
  }
  return retVal;
});
#endif
